#include "hip/hip_runtime.h"



#include "rainforest_function.h"
#include "lyra2/cuda_lyra2_vectors.h" 

//#include "math_functions.h" 


__device__ static void rfv2_update(rfv2_ctx_t *ctx, const void *msg, size_t len, uint64_t * __restrict__ RamBox)
{
	const uchar *msg8 = (const uchar *)msg;

	while (len > 0) {

		if (!(ctx->len & 3) && len >= 4) {
			ctx->word = *(uint *)msg8;
			ctx->len += 4;
			rfv2_one_round(ctx,RamBox);
			msg8 += 4;
			len -= 4;
			continue;
		}
/*
		ctx->word |= ((uint)*msg8++) << (8 * (ctx->len++ & 3));
		len--;
		if (!(ctx->len & 3))
			rfv2_one_round(ctx,RamBox);
*/
	}  
} 

__device__ static inline void rfv2_pad256(rfv2_ctx_t *ctx, uint64_t * __restrict__ RamBox)
{
	const uchar pad256[32] = { 0, };
	uint pad; 

	pad = (32 - ctx->len) & 0xF;
	if (pad)
		rfv2_update(ctx, pad256, pad,  RamBox);   
}  

__device__ static void rfv2_final( rfv2_ctx_t *ctx, uint64_t * __restrict__ RamBox)
{ 

	rfv2_one_round(ctx, RamBox);
	rfv2_one_round(ctx, RamBox);     

/*
	rfv2_one_round(ctx, RamBox);
	rfv2_one_round(ctx, RamBox);
	rfv2_one_round(ctx, RamBox); 
*/
}

__device__ static uint32_t sin_scaled(uint x)
{
	int i;

	i = ((x * 42722829) >> 24) - 128;
	x = 15 * i * i * abs(i);  // 0 to 15<<21
	x = (x + (x >> 4)) >> 17;
	return 257 - x;
}
  
////////////////////////// equivalent of rfv2_cpuminer.c ////////////////////////
#define Looping 256         
#define TPB 64 
#define TPB2 512 
__global__ __launch_bounds__(TPB, 4)
void rf256v2_init_gpu(uint32_t thr_id, uint32_t threads, uint32_t startNounce, uint32_t * __restrict__ zCounter, uint32_t * __restrict__ zSelected)
{

	uint event_thread = (blockDim.x * blockIdx.x + threadIdx.x);

	uint32_t data[20];

	((uint16 *)data)[0] = ((const uint16 *)pData)[0];
	((uint4 *)data)[4] = ((const uint4 *)pData)[4];

for (int j = 0; j < Looping; j++) {
	uint32_t NonceIterator = cuda_swab32(startNounce + Looping * event_thread + j);
	data[19] = NonceIterator;
    uint32_t count;

	uint32_t msgh;

	msgh = rf_crc32_mem(0, (uint8_t*)data, 80);

	uint32_t loops = sin_scaled(msgh);

	if (loops == 2) { count = atomicAdd(&zCounter[0], 1); zSelected[count] = NonceIterator; }
}
}



__global__ __launch_bounds__(TPB2, 4)
void rf256v2_hash_gpu(uint32_t thr_id, uint32_t threads, uint32_t startNounce, uint32_t *   output, 
uint64_t * __restrict__ DieRambox, uint32_t * __restrict__ zCounter, uint32_t * __restrict__ zSelected)
{ 
 
	const uint32_t rfv2_iv[8] = { 0xd390e978,  0x7b9bc8b3,  0x6e86c40a,  0x6bb3384e,  0xed7c6833,  0x0a4b3573,  0x774c2597,  0x1b61aa7a };
	uint event_thread = (blockDim.x * blockIdx.x + threadIdx.x);


if (event_thread <= threads) {

		uint64_t * __restrict__ RamBox = &DieRambox[0];
		rfv2_ctx_t ctx; 
		uint32_t data[20];

		uint32_t NonceIterator = __ldg(&zSelected[event_thread]);//cuda_swab32(startNounce + event_thread);
		((uint16 *)data)[0] = ((const uint16 *)pData)[0];
		((uint4 *)data)[4] = ((const uint4 *)pData)[4];
		
		data[19] = NonceIterator;


		uint loop, loops; 
		uint msgh;  

	((uint8*)ctx.hash.d)[0] = ((const uint8 *)rfv2_iv)[0];
	
	ctx.crc = RFV2_INIT_CRC;  
	ctx.word = ctx.len = 0; 

	ctx.rb_o = 0;       
	ctx.rb_l = RFV2_RAMBOX_SIZE/2; 

	msgh = rf_crc32_mem(0, (uint8_t*)data, 80);
	ctx.rb_o = msgh % ctx.rb_l; 
	ctx.rb_l = (ctx.rb_l - ctx.rb_o) * 2;

	loops = 2;


	ctx.left_bits =  0;

	for (loop = 0; loop < loops; loop++) {      
		rfv2_update(&ctx, (uint8_t*)data, 80, RamBox);                         
		// pad to the next 256 bit boundary 
		rfv2_pad256(&ctx, RamBox);
	} 
 
	rfv2_final( &ctx, RamBox);   

	uint64_t Sol = MAKE_ULONGLONG(ctx.hash.d[3], ctx.hash.d[4]);


	if (Sol <= ((uint64_t*)pTarget)[3]) {

		atomicMin(&output[0], cuda_swab32(NonceIterator));
	}

 }
}




__host__
void rainforest_init(int thr_id, uint32_t threads, const void *box)
{  
threads =  threads;
//	hipSetDevice(device_map[thr_id]);
	// just assign the device pointer allocated in main loop

	//	hipMemcpyToSymbol(HIP_SYMBOL(GYLocal),&hash1[thr_id], 8 * sizeof(uint32_t) * threads);
	//	hipMalloc((void**)&GYLocal[thr_id], 8 * sizeof(uint32_t) * threads);
uint32_t aggr_size =(uint32_t) (RFV2_RAMBOX_SIZE/AGGR);
CUDA_SAFE_CALL(hipMalloc((void**)&Selected[thr_id], threads * sizeof(uint32_t)));

CUDA_SAFE_CALL(hipMalloc((void**)&Counter[thr_id], sizeof(uint32_t)));

CUDA_SAFE_CALL(hipMalloc((void**)&TheRamBox[thr_id], 1 * RFV2_RAMBOX_SIZE * sizeof(uint64_t)));


CUDA_SAFE_CALL(hipMalloc(&d_aMinNonces[thr_id], 2 * sizeof(uint32_t)));
CUDA_SAFE_CALL(hipHostMalloc(&h_aMinNonces[thr_id], 2 * sizeof(uint32_t)));



uint64_t *Boxptr1 = &TheRamBox[thr_id][0];
CUDA_SAFE_CALL(hipMemcpyAsync(Boxptr1, box, RFV2_RAMBOX_SIZE * sizeof(uint64_t), hipMemcpyHostToDevice));


}
  
 


__host__
void rainforest_setBlockTarget(int thr_id, int throughput, const void* pDataIn, const void *pTargetIn)
{
	//	hipSetDevice(device_map[thr_id]);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pData), pDataIn, 80, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}


__host__
uint32_t rainforest_cpu_hash(int thr_id, uint32_t threadu, uint32_t startNounce)
{
//	hipSetDevice(device_map[thr_id]);
	uint32_t TheCounter[1];
	uint32_t result[1] ={ UINT32_MAX};
	CUDA_SAFE_CALL(hipMemset(d_aMinNonces[thr_id], 0xff, 2* sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMemset(h_aMinNonces[thr_id], 0xff, 2 *sizeof(uint32_t)));

	CUDA_SAFE_CALL(hipMemset(Counter[thr_id], 0x0,  sizeof(uint32_t)));

//	CUDA_SAFE_CALL(hipMemset(h_MinNonces[thr_id], 0xff, sizeof(uint32_t)));
//	int dev_id = device_map[thr_id % MAX_GPUS];
	uint32_t threads = threadu/ Looping;
	uint32_t tpb = TPB;
	uint32_t tpb2 = TPB2;
	dim3 gridyloop1(threads / tpb);
	dim3 blockyloop1(tpb);


	rf256v2_init_gpu << < gridyloop1, blockyloop1 >> >(thr_id, threads, startNounce,Counter[thr_id],Selected[thr_id]);


	CUDA_SAFE_CALL(hipMemcpy(TheCounter, Counter[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));

	dim3 gridyloop((TheCounter[0] + tpb2 -1) / tpb2);
	dim3 blockyloop(tpb2);
	hipDeviceSynchronize();

	rf256v2_hash_gpu << < gridyloop, blockyloop >> >(thr_id, TheCounter[0], startNounce, d_aMinNonces[thr_id],TheRamBox[thr_id], Counter[thr_id], Selected[thr_id]);

	CUDA_SAFE_CALL(hipMemcpy(h_aMinNonces[thr_id], d_aMinNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));
//	CUDA_SAFE_CALL(hipMemset(d_aMinNonces[thr_id], 0xff, sizeof(uint32_t)));
//	CUDA_SAFE_CALL(hipMemcpy(result, d_aMinNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	result[0] = h_aMinNonces[thr_id][0];
	return result[0];

}
