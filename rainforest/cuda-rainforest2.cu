#include "hip/hip_runtime.h"



#include "rainforest_function.h"
#include "lyra2/cuda_lyra2_vectors.h" 

//#include "math_functions.h" 


__device__ static void rfv2_update(rfv2_ctx_t *ctx, const void *msg, size_t len, uint64_t * __restrict__ RamBox)
{
	const uchar *msg8 = (const uchar *)msg;

	while (len > 0) {

		if (!(ctx->len & 3) && len >= 4) {
			ctx->word = *(uint *)msg8;
			ctx->len += 4;
			rfv2_one_round(ctx,RamBox);
			msg8 += 4;
			len -= 4;
			continue;
		}
/*
		ctx->word |= ((uint)*msg8++) << (8 * (ctx->len++ & 3));
		len--;
		if (!(ctx->len & 3))
			rfv2_one_round(ctx,RamBox);
*/
	}  
} 

__device__ static inline void rfv2_pad256(rfv2_ctx_t *ctx, uint64_t * __restrict__ RamBox)
{
	const uchar pad256[32] = { 0, };
	uint pad; 

	pad = (32 - ctx->len) & 0xF;
	if (pad)
		rfv2_update(ctx, pad256, pad,  RamBox);   
}  

__device__ static void rfv2_final( rfv2_ctx_t *ctx, uint64_t * __restrict__ RamBox)
{ 

	rfv2_one_round(ctx, RamBox);
	rfv2_one_round(ctx, RamBox);     

/*
	rfv2_one_round(ctx, RamBox);
	rfv2_one_round(ctx, RamBox);
	rfv2_one_round(ctx, RamBox); 
*/
}

__device__ static uint32_t sin_scaled(uint x)
{
	int i;

	i = ((x * 42722829) >> 24) - 128;
	x = 15 * i * i * abs(i);  // 0 to 15<<21
	x = (x + (x >> 4)) >> 17;
	return 257 - x;
}
  
////////////////////////// equivalent of rfv2_cpuminer.c ////////////////////////
         
#define TPB 64 
__global__ __launch_bounds__(TPB, 8)
void rf256v2_hash_gpu(uint32_t thr_id, uint32_t threads, uint32_t startNounce, uint32_t *   output, uint64_t * __restrict__ DieRambox, uint16_t * __restrict__ DieIndex)
{ 
 
	const uint32_t rfv2_iv[8] = { 0xd390e978,  0x7b9bc8b3,  0x6e86c40a,  0x6bb3384e,  0xed7c6833,  0x0a4b3573,  0x774c2597,  0x1b61aa7a };
	uint event_thread = (blockDim.x * blockIdx.x + threadIdx.x);
 
		uint64_t * __restrict__ RamBox = &DieRambox[0];
		rfv2_ctx_t ctx; 
		uint32_t data[20];
		uint32_t NonceIterator = cuda_swab32(startNounce + event_thread);
		((uint16 *)data)[0] = ((const uint16 *)pData)[0];
		((uint4 *)data)[4] = ((const uint4 *)pData)[4];
		data[19] = NonceIterator;
 
	uint loop, loops; 
	uint msgh;  

	((uint8*)ctx.hash.d)[0] = ((const uint8 *)rfv2_iv)[0];
	
	ctx.crc = RFV2_INIT_CRC;  
	ctx.word = ctx.len = 0; 
	ctx.changes = 0;    
	ctx.gchanges = 0;
	ctx.rb_o = 0;       
	ctx.rb_l = RFV2_RAMBOX_SIZE/2; 
	ctx.LocalIndex = &DieIndex[RFV2_RAMBOX_SIZE*event_thread/ AGGR];

	msgh = rf_crc32_mem(0, (uint8_t*)data, 80);
	ctx.rb_o = msgh % ctx.rb_l; 
	ctx.rb_l = (ctx.rb_l - ctx.rb_o) * 2;

	loops = sin_scaled(msgh);  

	ctx.left_bits = (loops >= 128)? 4 : (loops >= 64) ? 3 : (loops >= 32) ? 2 : (loops >= 16) ? 1 : 0;


/*  
	if (event_thread == 1) 
	{
	printf("rb_o = %08x rb_l = %08x \n", ctx.rb_o, ctx.rb_l);
	printf("event_thread = %d msgh = %08x loops = %d\n",event_thread,msgh,loops);
	}
*/
	for (loop = 0; loop < loops; loop++) {      
		rfv2_update(&ctx, (uint8_t*)data, 80, RamBox);                         
		// pad to the next 256 bit boundary 
		rfv2_pad256(&ctx, RamBox);
	} 
 
	rfv2_final( &ctx, RamBox);  

	uint64_t Sol = MAKE_ULONGLONG(ctx.hash.d[3], ctx.hash.d[4]);


	if (Sol <= ((uint64_t*)pTarget)[3]) {
//	if (ctx.hash.q[3] <= ((uint64_t*)pTarget)[3]) {

/*
		printf("GPU hash  %08x %08x %08x %08x   %08x %08x %08x %08x   \n", ctx.hash.d[0], ctx.hash.d[1], ctx.hash.d[2], ctx.hash.d[3],
			ctx.hash.d[4], ctx.hash.d[5], ctx.hash.d[6], ctx.hash.d[7]);
	printf("GPU number of changes %d global changes %d\n", ctx.changes, ctx.gchanges);
*/
		atomicMin(&output[0], cuda_swab32(NonceIterator));
	}
 
	for (int i = 0; i<ctx.changes; i++)
		ctx.LocalIndex[ctx.hist[i]/AGGR] = 0;

}




__host__
void rainforest_init(int thr_id, uint32_t threads, const void *box)
{  
//	hipSetDevice(device_map[thr_id]);
	// just assign the device pointer allocated in main loop

	//	hipMemcpyToSymbol(HIP_SYMBOL(GYLocal),&hash1[thr_id], 8 * sizeof(uint32_t) * threads);
	//	hipMalloc((void**)&GYLocal[thr_id], 8 * sizeof(uint32_t) * threads);
uint32_t aggr_size =(uint32_t) (RFV2_RAMBOX_SIZE/AGGR);
CUDA_SAFE_CALL(hipMalloc((void**)&TheRamBox[thr_id], 1 * RFV2_RAMBOX_SIZE * sizeof(uint64_t)));
CUDA_SAFE_CALL(hipMalloc((void**)&TheIndex[thr_id],  threads * aggr_size * sizeof(uint16_t)));
CUDA_SAFE_CALL(hipMalloc(&d_aMinNonces[thr_id], 2 * sizeof(uint32_t)));
CUDA_SAFE_CALL(hipHostMalloc(&h_aMinNonces[thr_id], 2 * sizeof(uint32_t)));

uint16_t *TheCarry = (uint16_t*)calloc(threads * aggr_size, sizeof(uint16_t));

uint64_t *Boxptr1 = &TheRamBox[thr_id][0];
CUDA_SAFE_CALL(hipMemcpyAsync(Boxptr1, box, RFV2_RAMBOX_SIZE * sizeof(uint64_t), hipMemcpyHostToDevice));


uint16_t *Boxptr = &TheIndex[thr_id][0];
CUDA_SAFE_CALL(hipMemcpyAsync(Boxptr, TheCarry, threads * aggr_size * sizeof(uint16_t), hipMemcpyHostToDevice));
free(TheCarry);


//	hipMalloc(&Header[thr_id], sizeof(uint32_t) * 8); 
//	hipMalloc(&buffer_a[thr_id], 4194304 * 64);
}
  
 


__host__
void rainforest_setBlockTarget(int thr_id, int throughput, const void* pDataIn, const void *pTargetIn)
{
	//	hipSetDevice(device_map[thr_id]);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pData), pDataIn, 80, 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}


__host__
uint32_t rainforest_cpu_hash(int thr_id, uint32_t threads, uint32_t startNounce)
{
//	hipSetDevice(device_map[thr_id]);
	uint32_t result[1] ={ UINT32_MAX};
	CUDA_SAFE_CALL(hipMemset(d_aMinNonces[thr_id], 0xff, 2* sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMemset(h_aMinNonces[thr_id], 0xff, 2 *sizeof(uint32_t)));
//	CUDA_SAFE_CALL(hipMemset(h_MinNonces[thr_id], 0xff, sizeof(uint32_t)));
//	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB;

	dim3 gridyloop(threads / tpb);
	dim3 blockyloop(tpb);

	rf256v2_hash_gpu << < gridyloop, blockyloop >> >(thr_id, threads, startNounce, d_aMinNonces[thr_id],TheRamBox[thr_id],TheIndex[thr_id]);

	CUDA_SAFE_CALL(hipMemcpy(h_aMinNonces[thr_id], d_aMinNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));
//	CUDA_SAFE_CALL(hipMemset(d_aMinNonces[thr_id], 0xff, sizeof(uint32_t)));
//	CUDA_SAFE_CALL(hipMemcpy(result, d_aMinNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));

	result[0] = h_aMinNonces[thr_id][0];
	return result[0];

}
